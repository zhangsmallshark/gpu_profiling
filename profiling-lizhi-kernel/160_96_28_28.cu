
#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#define TH 3
#define TW 2
#define N1 160
#define N2 96
#define N3 128
#define Channel 1
#define TC 8
#define TCS ((N1-1)/TC + 1)
#define Height 28
#define Width 28
#define R 3
#define S 3
#define THS ((Height-1)/TH + 1)
#define TWS ((Width-1)/TW+1)
#define K_SIZE (N1*Channel)
#if TC > (TH*TW)
#define REGISTER_BUFFER_SIZE (TC)
#else
#define REGISTER_BUFFER_SIZE (TH*TW)
#endif
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class Conv{
public:
    unsigned int H;
    unsigned int W;
    unsigned int C;
    unsigned int B;
    unsigned int N;
    unsigned int PAD;
    unsigned int hOut;
    unsigned int wOut;
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                    unsigned int pad,unsigned int r,unsigned int s,unsigned int stride);
    float *forward(float *input);
};
void Conv::initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                      unsigned int pad,unsigned int r,unsigned int s,unsigned int stride){
    this->B = b;
    this->C = c;
    this->H = h;
    this->W = w;
    this->N = n;
    this->hOut = (H+2*pad - r)/stride + 1;
    this->wOut = (W+2*pad - s)/stride + 1;
    hipMalloc(&kernel,sizeof(float)*C*N*r*s);
    hipMalloc(&this->output,sizeof(float)*B*hOut*wOut*N);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NHWC,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/r,
            /*kernel_width=*/s);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/pad,
            /*pad_width=*/pad,
            /*vertical_stride=*/stride,
            /*horizontal_stride=*/stride,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NHWC,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/N,
            /*image_height=*/hOut,
            /*image_width=*/wOut);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = r*s*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,r*s*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * Conv::forward(float *input) {
    hipMemset(output, 0, B*N*hOut*wOut*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<R;++r){
        for(unsigned int s=0;s<S;++s){
            data_array[r*S+s] = kernel[c_index*N2*9+r*3*N2+s*N2+n_id];
        }
    }
}
__device__ void switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
	switch (switch_condition) {
		case 0:
			for ( int r = 0; r < 1; r++) {
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(0-s)] += result;
				}
			}
		break;
		case 1:
			for ( int r = 0; r < 1; r++) {
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(1-s)] += result;
				}
			}
		break;
		case 2:
			for ( int r = 0; r < 1; r++) {
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(2-s)] += result;
				}
			}
		break;
		case 3:
			for ( int r = 0; r < 1; r++) {
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(3-s)] += result;
				}
			}
		break;
		case 4:
			for ( int r = 0; r < 2; r++) {
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(0-s)] += result;
				}
			}
		break;
		case 5:
			for ( int r = 0; r < 2; r++) {
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(1-s)] += result;
				}
			}
		break;
		case 6:
			for ( int r = 0; r < 2; r++) {
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(2-s)] += result;
				}
			}
		break;
		case 7:
			for ( int r = 0; r < 2; r++) {
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(3-s)] += result;
				}
			}
		break;
		case 8:
			for ( int r = 0; r < 3; r++) {
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(0-s)] += result;
				}
			}
		break;
		case 9:
			for ( int r = 0; r < 3; r++) {
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(1-s)] += result;
				}
			}
		break;
		case 10:
			for ( int r = 0; r < 3; r++) {
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(2-s)] += result;
				}
			}
		break;
		case 11:
			for ( int r = 0; r < 3; r++) {
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(3-s)] += result;
				}
			}
		break;
		case 12:
			for ( int r = 1; r < 3; r++) {
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(0-s)] += result;
				}
			}
		break;
		case 13:
			for ( int r = 1; r < 3; r++) {
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(1-s)] += result;
				}
			}
		break;
		case 14:
			for ( int r = 1; r < 3; r++) {
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(2-s)] += result;
				}
			}
		break;
		case 15:
			for ( int r = 1; r < 3; r++) {
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(3-s)] += result;
				}
			}
		break;
		case 16:
			for ( int r = 2; r < 3; r++) {
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*2+(0-s)] += result;
				}
			}
		break;
		case 17:
			for ( int r = 2; r < 3; r++) {
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*2+(1-s)] += result;
				}
			}
		break;
		case 18:
			for ( int r = 2; r < 3; r++) {
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*2+(2-s)] += result;
				}
			}
		break;
		case 19:
			for ( int r = 2; r < 3; r++) {
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*2+(3-s)] += result;
				}
			}
		break;

	}
}
__global__ void transform(float *matrix, float *matrix2){
    for(unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;global_id<N1*Height*Width;global_id+=gridDim.x * blockDim.x){
        const float v = matrix[global_id];
        unsigned int c = global_id % N1;
        unsigned int hw = global_id / N1;
        int h = (hw)/Width+1;
        int w = (hw)%Width+1;
        int th_start = min(h/TH,THS-1);
        int tw_start = min(w/TW,TWS-1);
        for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
            if((tile_h_id*TH+TH+2)<=h){
                break;
            }
            for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
                if((tile_w_id*TW+TW+2)<=w){
                    break;
                }
                unsigned int tile_id = tile_h_id * TWS + tile_w_id;
                unsigned int abs_h = h - tile_h_id*TH;
                unsigned int abs_w = w - tile_w_id*TW;
                matrix2[c*THS*TWS*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+abs_h*(TW+2)+abs_w] = v;
            }
        }
    }

}
__device__ void load_input_2_shared_memory(float *values,float *shared_input,unsigned int warp_id,unsigned int lane_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<TC&&tile_c_id+c_id<N1;c_id+=blockDim.x/32){
        for(unsigned int id = lane_id;id<(TH+2)*(TW+2);id+=32){
            shared_input[c_id*(TH+2)*(TW+2)+id] = values[(tile_c_id+c_id)*(THS*TWS)*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+id];
        }
    }
}
__global__ void conv2d(float * __restrict__ values,const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[TC*(TH+2)*(TW+2)];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / (THS * TWS);
    const unsigned int th_id = (tile_id - tc_id * (THS*TWS))/TWS;
    const unsigned int tw_id = (tile_id - tc_id * (THS*TWS))%TWS;
    const unsigned int h_start = th_id * TH;
    const unsigned int w_start = tw_id * TW;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    load_input_2_shared_memory(values,input,warp_id,lane_id,tile_id - tc_id * (THS*TWS),tc_id*TC);
    __syncthreads();
    float v;
    unsigned int n = threadIdx.x;
    for(unsigned int c=tc_id*TC;c<tc_id*TC+TC;c++){
        load_data_2_register(data_array,c,kernel,n);
        unsigned int abs_c = c - tc_id * TC;
        for(unsigned int i=0;i<(TH+2)*(TW+2);++i){
            v = input[i + abs_c*(TH+2)*(TW+2)];
            if(v == 0.0f){
                continue;
            }
            switch_function(i,data_array,v,temp_result);
        }
    }
    for (unsigned int th = 0; th < TH; ++th) {
        for (unsigned int tw = 0; tw < TW; ++tw) {
            if (h_start + th >= Height || w_start + tw >= Width) {
                continue;
            }
            atomicAdd(&outputs[(h_start + th) * Width * N2 + (w_start + tw) * N2 +n],temp_result[(th * TW + tw)]);
        }
    }
}
float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[N1*Height*Width];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<N1*Height*Width;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,N1*Height*Width*sizeof(float));
    hipMemcpy(device_input,input,N1*Height*Width*sizeof(float),hipMemcpyHostToDevice);
    float *K2 = new float[N1*N2*9];
    for(int i=0;i<N1*N2*9;++i){
        K2[i] = 1.0f;
    }
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    float *device_K2;
    float *device_out;
    hipMalloc(&device_out,Height*Width*N2*sizeof(float));
    hipMemset(device_out,0,Height*Width*N2*sizeof(float));
    hipMalloc(&device_K2,N1*N2*9*sizeof(float));
    hipMemcpy(device_K2,K2,N1*N2*9*sizeof(float),hipMemcpyHostToDevice);
    float *matrix;
    hipMalloc(&matrix,N1*(TH+2)*(TW+2)*THS*TWS*sizeof(float));

    hipEventRecord(event_start);
    hipMemset(matrix,0,N1*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    transform<<<108,512>>>(device_input,matrix);
    conv2d<<<TCS*THS*TWS,N2>>>(matrix,device_K2, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time3;
    hipEventElapsedTime(&time3, event_start, event_stop);
    cout<<N2<<","<<N1<<","<<Height<<","<<Width<<","<<TH<<","<<TW<<","<<TC<<","<<time3<<endl;
    return 0;
}

