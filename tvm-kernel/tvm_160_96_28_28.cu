//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[768];
  __shared__ float kernel_shared[1728];
  float pad_temp_shared_local[2];
  float kernel_shared_local[8];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 10; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 19) & 15)))) && (((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 19) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + ((((int)threadIdx.x) * 19) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 1))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 1) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 1) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 2))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 2) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 2) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 3))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 3) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 3) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 4))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 4) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 4) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 5))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 5) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 5) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 6))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 6) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 6) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 6) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 7))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 7) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 7) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 7) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 8))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 8) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 8) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 8) & 15)) - 29))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 759) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 9))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 9) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 9) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 9) & 15)) - 29))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 758) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 10))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 10) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 10) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 10) & 15)) - 29))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 757) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 11))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 11) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 11) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 11) & 15)) - 29))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 756) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 12))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 12) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 12) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 12) & 15)) - 29))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 755) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 13))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 13) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 13) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 13) & 15)) - 29))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 754) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 14))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 14) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 14) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 14) & 15)) - 29))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 753) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 15))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 15) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 15) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 15) & 15)) - 29))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) < 47) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 752) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 16))] = (((((1 <= (((int)blockIdx.y) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3))) && ((((int)blockIdx.y) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 19) & 15)))) && (((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 19) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) / 3) * 784)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3) * 28)) + (((int)blockIdx.x) * 14)) + ((((int)threadIdx.x) * 19) & 15)) - 29))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 751) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 17))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 1) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 1) & 15)) - 29))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 750) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 18))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 19) + 2) & 15)) < 29)) ? data[((((((((rc_outer * 12544) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) / 3) * 784)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 19) + 2) & 15)) - 29))] : 0.000000e+00f);
        }
      }
    }
    kernel_shared[(((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + (((((int)threadIdx.x) * 14) / 48) * 1440)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 14) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 1))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + (((((int)threadIdx.x) * 14) / 48) * 1440)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 14) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 2))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + (((((int)threadIdx.x) * 14) / 48) * 1440)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 14) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 3))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 1) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 1) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 4))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 1) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 1) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 5))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 1) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 1) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 6))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 2) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 2) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 7))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 2) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 2) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 8))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 2) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 2) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 9))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 3) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 3) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 10))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 3) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 3) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 11))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 3) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 3) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 12))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 4) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 4) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 13))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 4) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 4) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 14))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 4) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 4) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 15))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 5) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 5) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 16))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 5) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 5) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 17))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 5) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 5) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 18))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 6) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 6) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 19))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 6) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 6) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 20))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 6) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 6) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 21))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 7) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 7) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 22))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 7) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 7) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 23))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 7) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 7) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 24))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 8) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 8) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 25))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 8) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 8) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 26))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 8) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 8) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 27))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 9) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 9) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 28))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 9) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 9) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 29))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 9) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 9) % 48) * 3)) + 2))];
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 10) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + (((((int)threadIdx.x) * 14) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 566) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1698) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 30))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 10) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 10) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 10) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + (((((int)threadIdx.x) * 14) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 566) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1697) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 31))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 10) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 10) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 10) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + (((((int)threadIdx.x) * 14) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 566) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1696) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 32))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 10) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 10) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 11) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + (((((int)threadIdx.x) * 14) + 11) / 3)) < 192) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 565) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1695) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 33))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 11) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 11) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 11) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + (((((int)threadIdx.x) * 14) + 11) / 3)) < 192) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 565) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1694) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 34))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 11) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 11) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 11) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + (((((int)threadIdx.x) * 14) + 11) / 3)) < 192) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 565) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1693) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 35))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 11) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 11) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 12) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + ((((int)threadIdx.x) * 14) / 3)) < 188) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 564) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1692) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 36))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 12) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 12) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 12) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + ((((int)threadIdx.x) * 14) / 3)) < 188) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 564) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1691) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 37))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 12) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 12) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 12) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + ((((int)threadIdx.x) * 14) / 3)) < 188) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 564) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1690) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 38))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 12) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 12) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 13) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + (((((int)threadIdx.x) * 14) + 13) / 3)) < 192) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 563) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1689) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 39))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 13) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 13) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 13) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + (((((int)threadIdx.x) * 14) + 13) / 3)) < 192) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 563) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1688) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 40))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 13) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 13) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 14) + 13) / 48)) < 12) {
      if (((((int)threadIdx.z) * 64) + (((((int)threadIdx.x) * 14) + 13) / 3)) < 192) {
        if (((((int)threadIdx.z) * 192) + (((int)threadIdx.x) * 14)) < 563) {
          if (((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) < 1687) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 42)) + 41))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 5760)) + ((((((int)threadIdx.x) * 14) + 13) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 14) + 13) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 48))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 576))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 9))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 144))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 153))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 288))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 297))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 432))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 441))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 10))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 145))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 154))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 289))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 298))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 433))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 442))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 50))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 11))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 146))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 155))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 290))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 299))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 434))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 443))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 16))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 3))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 12))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 147))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 156))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 291))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 300))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 435))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 444))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 17))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 4))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 148))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 157))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 292))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 301))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 436))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 445))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 66))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 5))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 14))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 149))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 158))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 293))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 302))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 437))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 446))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 32))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 80))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 15))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 150))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 159))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 294))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 303))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 438))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 447))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 33))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 7))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 16))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 151))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 160))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 295))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 304))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 439))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 448))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 34))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 8))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 17))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 152))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 161))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 296))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 305))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 440))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 449))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 96))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 27))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 162))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 171))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 306))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 315))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 450))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 459))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 97))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 19))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 28))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 163))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 172))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 307))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 316))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 451))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 460))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 20))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 29))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 164))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 173))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 308))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 317))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 452))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 461))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 112))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 160))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 21))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 30))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 165))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 174))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 309))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 318))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 453))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 462))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 113))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 161))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 22))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 31))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 166))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 175))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 310))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 319))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 454))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 463))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 114))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 23))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 32))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 167))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 176))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 311))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 320))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 455))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 464))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 176))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 33))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 168))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 177))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 312))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 321))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 456))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 465))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 129))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 177))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 25))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 34))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 169))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 178))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 313))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 322))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 457))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 466))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 130))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 178))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 26))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 35))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 170))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 179))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 314))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 323))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 458))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 467))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 192))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 240))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 45))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 180))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 189))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 324))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 333))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 468))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 477))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 193))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 241))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 37))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 46))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 181))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 190))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 325))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 334))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 469))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 478))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 194))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 242))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 38))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 47))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 182))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 191))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 326))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 335))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 470))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 479))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 256))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 39))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 48))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 183))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 192))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 327))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 336))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 471))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 480))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 257))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 40))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 184))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 193))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 328))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 337))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 472))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 481))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 258))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 41))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 50))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 185))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 194))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 329))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 338))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 473))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 482))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 224))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 272))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 42))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 51))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 186))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 195))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 330))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 339))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 474))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 483))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 225))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 273))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 43))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 52))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 187))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 196))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 331))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 340))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 475))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 484))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 226))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 274))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 44))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 53))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 188))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 197))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 332))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 341))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 476))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 485))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 288))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 336))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 54))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 63))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 198))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 207))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 342))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 351))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 486))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 495))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 289))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 337))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 55))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 64))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 199))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 208))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 343))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 352))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 487))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 496))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 290))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 338))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 56))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 65))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 200))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 209))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 344))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 353))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 488))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 497))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 304))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 352))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 57))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 66))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 201))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 210))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 345))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 354))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 489))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 498))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 305))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 353))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 58))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 67))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 202))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 211))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 346))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 355))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 490))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 499))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 306))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 354))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 59))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 68))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 203))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 212))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 347))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 356))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 491))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 500))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 320))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 368))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 60))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 69))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 204))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 213))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 348))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 357))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 492))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 501))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 321))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 369))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 61))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 70))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 205))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 214))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 349))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 358))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 493))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 502))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 322))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 370))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 62))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 71))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 206))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 215))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 350))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 359))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 494))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 503))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 384))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 432))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 81))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 216))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 225))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 360))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 369))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 504))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 513))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 385))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 433))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 73))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 82))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 217))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 226))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 361))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 370))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 505))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 514))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 386))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 434))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 74))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 83))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 218))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 227))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 362))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 371))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 506))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 515))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 400))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 448))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 75))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 84))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 219))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 228))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 363))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 372))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 507))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 516))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 401))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 449))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 76))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 85))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 220))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 229))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 364))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 373))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 508))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 517))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 402))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 450))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 77))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 86))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 221))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 230))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 365))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 374))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 509))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 518))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 416))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 464))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 78))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 87))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 222))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 231))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 366))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 375))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 510))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 519))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 417))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 465))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 79))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 88))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 223))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 232))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 367))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 376))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 511))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 520))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 418))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 466))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 80))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 89))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 224))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 233))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 368))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 377))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 512))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 521))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 480))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 528))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 90))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 99))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 234))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 243))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 378))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 387))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 522))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 531))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 481))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 529))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 91))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 100))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 235))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 244))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 379))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 388))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 523))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 532))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 482))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 530))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 92))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 101))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 236))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 245))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 380))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 389))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 524))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 533))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 496))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 544))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 93))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 102))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 237))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 246))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 381))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 390))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 525))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 534))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 497))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 545))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 94))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 103))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 238))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 247))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 382))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 391))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 526))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 535))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 498))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 546))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 95))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 104))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 239))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 248))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 383))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 392))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 527))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 536))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 512))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 560))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 96))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 105))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 240))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 249))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 384))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 393))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 528))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 537))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 513))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 561))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 97))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 106))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 241))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 250))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 385))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 394))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 529))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 538))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 514))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 562))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 98))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 107))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 242))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 251))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 386))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 395))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 530))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 539))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 576))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 624))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 108))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 117))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 252))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 261))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 396))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 405))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 540))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 549))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 577))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 625))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 109))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 118))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 253))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 262))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 397))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 406))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 541))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 550))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 578))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 626))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 110))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 119))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 254))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 263))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 398))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 407))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 542))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 551))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 592))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 640))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 111))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 120))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 255))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 264))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 399))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 408))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 543))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 552))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 593))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 641))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 112))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 121))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 256))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 265))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 400))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 409))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 544))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 553))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 594))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 642))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 113))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 122))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 257))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 266))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 401))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 410))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 545))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 554))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 608))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 656))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 114))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 123))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 258))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 267))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 402))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 411))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 546))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 555))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 609))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 657))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 115))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 124))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 259))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 268))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 403))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 412))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 547))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 556))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 610))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 658))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 116))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 125))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 260))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 269))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 404))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 413))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 548))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 557))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 672))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 720))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 126))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 135))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 270))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 279))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 414))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 423))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 558))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 567))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 673))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 721))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 127))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 136))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 271))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 280))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 415))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 424))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 559))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 568))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 674))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 722))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 128))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 137))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 272))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 281))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 416))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 425))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 560))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 569))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 688))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 736))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 129))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 138))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 273))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 282))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 417))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 426))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 561))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 570))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 689))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 737))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 130))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 139))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 274))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 283))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 418))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 427))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 562))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 571))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 690))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 738))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 131))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 140))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 275))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 284))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 419))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 428))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 563))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 572))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 704))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 752))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 132))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 141))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 276))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 285))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 420))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 429))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 564))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 573))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 705))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 753))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 133))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 142))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 277))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 286))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 421))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 430))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 565))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 574))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 706))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 754))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + 134))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 576) + 143))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 576) + 278))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 576) + 287))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 576) + 422))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 576) + 431))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 576) + 566))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 576) + 575))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
  }
  compute[((((((((int)blockIdx.z) * 9408) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 9408) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 784))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 9408) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 1568))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 9408) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 2352))] = compute_local[(3)];
}

int main(int argc, char *argv[]){
    dim3 grid(2,28,8);
    dim3 block(14,1,3);
    int C = atoi(argv[1]);
    int H = atoi(argv[3]);
    int W = atoi(argv[4]);
    int N = atoi(argv[2]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}
