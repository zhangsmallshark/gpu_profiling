//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[432];
  __shared__ float kernel_shared[2880];
  float pad_temp_shared_local[6];
  float kernel_shared_local[12];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    for (int rx_outer = 0; rx_outer < 3; ++rx_outer) {
      __syncthreads();
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 432) {
        pad_temp_shared[(((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)))] = (((((1 <= (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9)) && ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) / 9) * 49)) + ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 431) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 430) {
        if (((int)threadIdx.y) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
        }
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 429) {
        if (((int)threadIdx.y) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
        }
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 428) {
        if (((int)threadIdx.y) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
        }
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 427) {
        if (((int)threadIdx.y) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
        }
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 426) {
        if (((int)threadIdx.y) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
        }
      }
      kernel_shared[(((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + (((((int)threadIdx.y) * 14) / 48) * 1728)) + (rc_outer * 432)) + (((((int)threadIdx.y) * 14) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 1))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + (((((int)threadIdx.y) * 14) / 48) * 1728)) + (rc_outer * 432)) + (((((int)threadIdx.y) * 14) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 2))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + (((((int)threadIdx.y) * 14) / 48) * 1728)) + (rc_outer * 432)) + (((((int)threadIdx.y) * 14) % 48) * 9)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 3))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 1) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 1) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 4))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 1) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 1) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 5))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 1) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 1) % 48) * 9)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 6))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 2) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 2) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 7))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 2) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 2) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 8))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 2) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 2) % 48) * 9)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 9))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 3) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 3) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 10))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 3) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 3) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 11))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 3) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 3) % 48) * 9)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 12))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 4) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 4) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 13))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 4) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 4) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 14))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 4) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 4) % 48) * 9)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 15))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 5) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 5) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 16))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 5) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 5) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 17))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 5) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 5) % 48) * 9)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 18))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 6) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 6) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 19))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 6) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 6) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 20))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 6) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 6) % 48) * 9)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 21))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 7) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 7) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 22))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 7) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 7) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 23))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 7) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 7) % 48) * 9)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 24))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 8) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 8) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 25))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 8) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 8) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 26))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 8) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 8) % 48) * 9)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 27))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 9) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 9) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 28))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 9) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 9) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 29))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 9) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 9) % 48) * 9)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 30))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 10) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 10) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 31))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 10) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 10) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 32))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 10) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 10) % 48) * 9)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 33))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 11) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 11) % 48) * 9)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 34))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 11) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 11) % 48) * 9)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 35))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 11) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 11) % 48) * 9)) + rx_outer) + 6))];
      if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 14) + 12) / 48)) < 20) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 14)) < 948) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) < 2844) {
            if (((int)threadIdx.y) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 36))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 12) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 12) % 48) * 9)) + rx_outer))];
            }
          }
        }
      }
      if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 14) + 12) / 48)) < 20) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 14)) < 948) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) < 2843) {
            if (((int)threadIdx.y) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 37))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 12) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 12) % 48) * 9)) + rx_outer) + 3))];
            }
          }
        }
      }
      if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 14) + 12) / 48)) < 20) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 14)) < 948) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) < 2842) {
            if (((int)threadIdx.y) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 38))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 12) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 12) % 48) * 9)) + rx_outer) + 6))];
            }
          }
        }
      }
      if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 14) + 13) / 48)) < 20) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 14)) < 947) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) < 2841) {
            if (((int)threadIdx.y) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 39))] = kernel[(((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 13) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 13) % 48) * 9)) + rx_outer))];
            }
          }
        }
      }
      if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 14) + 13) / 48)) < 20) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 14)) < 947) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) < 2840) {
            if (((int)threadIdx.y) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 40))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 13) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 13) % 48) * 9)) + rx_outer) + 3))];
            }
          }
        }
      }
      if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 14) + 13) / 48)) < 20) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 14)) < 947) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) < 2839) {
            if (((int)threadIdx.y) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + 41))] = kernel[((((((((((int)blockIdx.z) * 34560) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.y) * 14) + 13) / 48) * 1728)) + (rc_outer * 432)) + ((((((int)threadIdx.y) * 14) + 13) % 48) * 9)) + rx_outer) + 6))];
            }
          }
        }
      }
      __syncthreads();
      for (int rc_inner_outer = 0; rc_inner_outer < 24; ++rc_inner_outer) {
        pad_temp_shared_local[(0)] = pad_temp_shared[(((rc_inner_outer * 18) + ((int)threadIdx.y)))];
        pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 18) + ((int)threadIdx.y)) + 1))];
        pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 18) + ((int)threadIdx.y)) + 2))];
        pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 18) + ((int)threadIdx.y)) + 9))];
        pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 18) + ((int)threadIdx.y)) + 10))];
        pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 18) + ((int)threadIdx.y)) + 11))];
        kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)))];
        kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)) + 1))];
        kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)) + 2))];
        kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)) + 3))];
        kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)) + 4))];
        kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)) + 5))];
        kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)) + 144))];
        kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)) + 145))];
        kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)) + 146))];
        kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)) + 147))];
        kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)) + 148))];
        kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 6)) + 149))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
      }
    }
  }
  compute[(((((((int)blockIdx.z) * 980) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 7)) + ((int)blockIdx.x)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 980) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 7)) + ((int)blockIdx.x)) + 49))] = compute_local[(1)];
}

//grid=(7,1,5),  block=(1,1,32)
int main(int argc, char *argv[]){
    dim3 grid(7,1,8);
    dim3 block(1,7,10);
    int C = atoi(argv[1]);
    int H = atoi(argv[3]);
    int W = atoi(argv[4]);
    int N = atoi(argv[2]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}
