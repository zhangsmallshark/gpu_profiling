//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[28];
  __shared__ float pad_temp_shared[224];
  __shared__ float kernel_shared[192];
  float pad_temp_shared_local[42];
  float kernel_shared_local[24];
  #pragma unroll
  for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
    #pragma unroll
    for (int yy_c_init = 0; yy_c_init < 7; ++yy_c_init) {
      compute_local[(((ff_c_init * 7) + yy_c_init))] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 96; ++rc_outer) {
    #pragma unroll
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      #pragma unroll
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
        pad_temp_shared[((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= (((((int)blockIdx.y) * 7) + (((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 112) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 7) + (((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 112) >> 4)) + ry_outer) < 15)) && (1 <= ((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 15))) && (((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 15) < 15)) ? data[((((((((rc_outer * 392) + (((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 112) * 196)) + (((int)blockIdx.y) * 98)) + ((((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 112) >> 4) * 14)) + (ry_outer * 14)) + ((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 15)) - 15))] : 0.000000e+00f);
      }
      #pragma unroll
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
        if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 6)) < 32) {
          if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 3)) < 64) {
            if ((((((int)threadIdx.z) * 24) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 192) {
              if (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 24) {
                kernel_shared[((((((int)threadIdx.z) * 24) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[((((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 6912)) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 6) * 1728)) + (rc_outer * 18)) + (((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) % 6) / 3) * 9)) + (ry_outer * 3)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) % 3)))];
              }
            }
          }
        }
      }
      __syncthreads();
      #pragma unroll
      for (int ax1 = 0; ax1 < 2; ++ax1) {
        #pragma unroll
        for (int ax2 = 0; ax2 < 7; ++ax2) {
          #pragma unroll
          for (int ax3 = 0; ax3 < 3; ++ax3) {
            pad_temp_shared_local[((((ax1 * 21) + (ax2 * 3)) + ax3))] = pad_temp_shared[(((((ax1 * 112) + (ax2 * 16)) + ax3) + ((int)threadIdx.x)))];
          }
        }
      }
      #pragma unroll
      for (int ax0 = 0; ax0 < 4; ++ax0) {
        #pragma unroll
        for (int ax11 = 0; ax11 < 2; ++ax11) {
          #pragma unroll
          for (int ax31 = 0; ax31 < 3; ++ax31) {
            kernel_shared_local[((((ax0 * 6) + (ax11 * 3)) + ax31))] = kernel_shared[(((((((int)threadIdx.z) * 24) + (ax0 * 6)) + (ax11 * 3)) + ax31))];
          }
        }
      }
      #pragma unroll
      for (int rc_inner_inner = 0; rc_inner_inner < 2; ++rc_inner_inner) {
        #pragma unroll
        for (int rx_inner_inner = 0; rx_inner_inner < 3; ++rx_inner_inner) {
          #pragma unroll
          for (int ff_c = 0; ff_c < 4; ++ff_c) {
            #pragma unroll
            for (int yy_c = 0; yy_c < 7; ++yy_c) {
              compute_local[(((ff_c * 7) + yy_c))] = (compute_local[(((ff_c * 7) + yy_c))] + (pad_temp_shared_local[((((rc_inner_inner * 21) + (yy_c * 3)) + rx_inner_inner))] * kernel_shared_local[((((ff_c * 6) + (rc_inner_inner * 3)) + rx_inner_inner))]));
            }
          }
        }
      }
    }
  }
  #pragma unroll
  for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 4; ++ff_inner_inner_inner) {
    #pragma unroll
    for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 7; ++yy_inner_inner_inner) {
      compute[(((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (ff_inner_inner_inner * 196)) + (((int)blockIdx.y) * 98)) + (yy_inner_inner_inner * 14)) + ((int)threadIdx.x)))] = compute_local[(((ff_inner_inner_inner * 7) + yy_inner_inner_inner))];
    }
  }
}


int main(int argc, char *argv[]){
    dim3 grid(1,2,3);
    dim3 block(14,1,8);
    int C = atoi(argv[1]);
    int H = atoi(argv[3]);
    int W = atoi(argv[4]);
    int N = atoi(argv[2]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}
