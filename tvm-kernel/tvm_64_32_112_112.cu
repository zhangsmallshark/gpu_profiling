//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float pad_temp_shared[1600];
  __shared__ float kernel_shared[2304];
  float pad_temp_shared_local[64];
  float kernel_shared_local[48];
  #pragma unroll
  for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
    #pragma unroll
    for (int xx_c_init = 0; xx_c_init < 2; ++xx_c_init) {
      compute_local[(((ff_c_init * 2) + xx_c_init))] = 0.000000e+00f;
      compute_local[((((ff_c_init * 2) + xx_c_init) + 4))] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 13; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 5) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 10)) / 10)) < 16) {
        if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 5)) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 10)) < 160) {
          if (((((((int)threadIdx.z) * 200) + (((int)threadIdx.y) * 50)) + (((int)threadIdx.x) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 1600) {
            if ((((((int)threadIdx.y) * 50) + (((int)threadIdx.x) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 200) {
              if (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 50) {
                pad_temp_shared[(((((((int)threadIdx.z) * 200) + (((int)threadIdx.y) * 50)) + (((int)threadIdx.x) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 8) + (((((int)threadIdx.y) * 5) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 10)) % 10))) && (((((int)blockIdx.y) * 8) + (((((int)threadIdx.y) * 5) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 10)) % 10)) < 113)) && (1 <= ((((int)blockIdx.x) * 8) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 10)))) && (((((int)blockIdx.x) * 8) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 10)) < 113)) ? data[(((((((((rc_outer * 200704) + (((int)threadIdx.z) * 25088)) + ((((((int)threadIdx.y) * 5) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 10)) / 10) * 12544)) + (((int)blockIdx.y) * 896)) + ((((((int)threadIdx.y) * 5) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 10)) % 10) * 112)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 10)) - 113))] : 0.000000e+00f);
              }
            }
          }
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 18; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      kernel_shared[(((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 18)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 1152)) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 9)) >> 4) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 9)) & 15) * 9)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 % 9)))];
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 2; ++rc_inner_outer) {
      #pragma unroll
      for (int ry_inner_outer = 0; ry_inner_outer < 3; ++ry_inner_outer) {
        #pragma unroll
        for (int ax1 = 0; ax1 < 8; ++ax1) {
          #pragma unroll
          for (int ax3 = 0; ax3 < 4; ++ax3) {
            pad_temp_shared_local[(((ax1 * 4) + ax3))] = pad_temp_shared[(((((((rc_inner_outer * 800) + (ax1 * 100)) + (((int)threadIdx.y) * 10)) + (ry_inner_outer * 10)) + (((int)threadIdx.x) * 2)) + ax3))];
            pad_temp_shared_local[((((ax1 * 4) + ax3) + 32))] = pad_temp_shared[((((((((rc_inner_outer * 800) + (ax1 * 100)) + (((int)threadIdx.y) * 10)) + (ry_inner_outer * 10)) + (((int)threadIdx.x) * 2)) + ax3) + 40))];
          }
        }
        #pragma unroll
        for (int ax0 = 0; ax0 < 2; ++ax0) {
          #pragma unroll
          for (int ax11 = 0; ax11 < 8; ++ax11) {
            #pragma unroll
            for (int ax31 = 0; ax31 < 3; ++ax31) {
              kernel_shared_local[((((ax0 * 24) + (ax11 * 3)) + ax31))] = kernel_shared[(((((((((int)threadIdx.z) * 288) + (ax0 * 144)) + (rc_inner_outer * 72)) + (ax11 * 9)) + (ry_inner_outer * 3)) + ax31))];
            }
          }
        }
        #pragma unroll
        for (int rc_inner_inner = 0; rc_inner_inner < 8; ++rc_inner_inner) {
          #pragma unroll
          for (int rx_inner_inner = 0; rx_inner_inner < 3; ++rx_inner_inner) {
            #pragma unroll
            for (int ff_c = 0; ff_c < 2; ++ff_c) {
              #pragma unroll
              for (int xx_c = 0; xx_c < 2; ++xx_c) {
                compute_local[(((ff_c * 2) + xx_c))] = (compute_local[(((ff_c * 2) + xx_c))] + (pad_temp_shared_local[((((rc_inner_inner * 4) + xx_c) + rx_inner_inner))] * kernel_shared_local[((((ff_c * 24) + (rc_inner_inner * 3)) + rx_inner_inner))]));
                compute_local[((((ff_c * 2) + xx_c) + 4))] = (compute_local[((((ff_c * 2) + xx_c) + 4))] + (pad_temp_shared_local[(((((rc_inner_inner * 4) + xx_c) + rx_inner_inner) + 32))] * kernel_shared_local[((((ff_c * 24) + (rc_inner_inner * 3)) + rx_inner_inner))]));
              }
            }
          }
        }
      }
    }
  }
  #pragma unroll
  for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2; ++ff_inner_inner_inner) {
    #pragma unroll
    for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 2; ++xx_inner_inner_inner) {
      compute[(((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 25088)) + (ff_inner_inner_inner * 12544)) + (((int)blockIdx.y) * 896)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner))] = compute_local[(((ff_inner_inner_inner * 2) + xx_inner_inner_inner))];
      compute[((((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 25088)) + (ff_inner_inner_inner * 12544)) + (((int)blockIdx.y) * 896)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 448))] = compute_local[((((ff_inner_inner_inner * 2) + xx_inner_inner_inner) + 4))];
    }
  }
}

//grid=(7,2,8),  block=(2,1,4)
int main(int argc, char *argv[]){
    dim3 grid(14,14,2);
    dim3 block(4,4,8);
    int C = atoi(argv[1]);
    int H = atoi(argv[3]);
    int W = atoi(argv[4]);
    int N = atoi(argv[2]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}
