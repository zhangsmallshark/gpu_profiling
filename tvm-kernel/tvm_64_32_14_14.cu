//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[256];
  __shared__ float kernel_shared[1536];
  float pad_temp_shared_local[16];
  float kernel_shared_local[8];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      if (((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)) < 256) {
        if (((int)threadIdx.x) < 4) {
          pad_temp_shared[(((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)))] = (((((1 <= (((int)blockIdx.y) + ry_outer)) && ((((int)blockIdx.y) + ry_outer) < 15)) && (1 <= (((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)) & 15))) && ((((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)) >> 4) * 196)) + (((int)blockIdx.y) * 14)) + (ry_outer * 14)) + (((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)) & 15)) - 15))] : 0.000000e+00f);
        }
      }
      if (((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)) < 255) {
        if (((int)threadIdx.x) < 4) {
          pad_temp_shared[((((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)) + 1))] = (((((1 <= (((int)blockIdx.y) + ry_outer)) && ((((int)blockIdx.y) + ry_outer) < 15)) && (1 <= ((((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)) + 1) & 15))) && (((((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)) + 1) & 15) < 15)) ? data[(((((((rc_outer * 3136) + (((((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)) + 1) >> 4) * 196)) + (((int)blockIdx.y) * 14)) + (ry_outer * 14)) + ((((((int)threadIdx.z) * 8) + (((int)threadIdx.x) * 2)) + 1) & 15)) - 15))] : 0.000000e+00f);
        }
      }
      kernel_shared[(((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)))] = kernel[((((((((int)threadIdx.z) * 576) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) / 3) * 9)) + (ry_outer * 3)) + ((((int)threadIdx.x) * 7) % 3)))];
      kernel_shared[((((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) + 1))] = kernel[((((((((int)threadIdx.z) * 576) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) / 3) * 9)) + (ry_outer * 3)) + (((((int)threadIdx.x) * 7) + 1) % 3)))];
      kernel_shared[((((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) + 2))] = kernel[((((((((int)threadIdx.z) * 576) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) / 3) * 9)) + (ry_outer * 3)) + (((((int)threadIdx.x) * 7) + 2) % 3)))];
      kernel_shared[((((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) + 3))] = kernel[(((((((((int)threadIdx.z) * 576) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) / 3) * 9)) + (ry_outer * 3)) + ((((int)threadIdx.x) * 7) % 3)) + 9))];
      kernel_shared[((((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) + 4))] = kernel[((((((((int)threadIdx.z) * 576) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) / 3) * 9)) + (ry_outer * 3)) + (((((int)threadIdx.x) * 7) + 1) % 3)))];
      kernel_shared[((((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) + 5))] = kernel[((((((((int)threadIdx.z) * 576) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) / 3) * 9)) + (ry_outer * 3)) + (((((int)threadIdx.x) * 7) + 2) % 3)))];
      if (((((((int)threadIdx.x) * 7) + 6) / 48) + ((int)threadIdx.z)) < 32) {
        if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 7) / 3)) < 510) {
          if (((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) < 1530) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) + 6))] = kernel[(((((((((int)threadIdx.z) * 576) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) / 3) * 9)) + (ry_outer * 3)) + ((((int)threadIdx.x) * 7) % 3)) + 18))];
            }
          }
        }
      }
      __syncthreads();
      pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 16))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 23))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 32))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 39))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 48))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 71))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 80))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 87))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 96))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 103))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 112))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
      kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 48))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 48) + 3))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 48) + 6))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 48) + 9))];
      kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 48) + 12))];
      kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 48) + 15))];
      kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 48) + 18))];
      kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 48) + 21))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(3)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(4)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(5)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(7)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 8))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 17))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 24))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 33))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 40))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 49))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 88))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 97))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 104))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 113))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 120))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 48) + 1))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 48) + 4))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 48) + 7))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 48) + 10))];
      kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 48) + 13))];
      kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 48) + 16))];
      kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 48) + 19))];
      kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 48) + 22))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(3)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(4)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(5)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(7)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 25))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 34))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 41))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 50))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 57))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 66))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 89))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 105))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 114))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 121))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 48) + 2))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 48) + 5))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 48) + 8))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 48) + 11))];
      kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 48) + 14))];
      kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 48) + 17))];
      kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 48) + 20))];
      kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 48) + 23))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(3)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(4)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(5)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(7)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 151))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 160))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 167))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 176))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 183))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 192))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 215))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 224))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 231))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 240))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 247))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 48) + 24))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 48) + 27))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 48) + 30))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 48) + 33))];
      kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 48) + 36))];
      kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 48) + 39))];
      kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 48) + 42))];
      kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 48) + 45))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(3)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(4)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(5)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(7)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 129))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 152))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 161))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 168))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 177))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 184))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 193))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 216))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 225))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 232))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 241))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 248))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 48) + 25))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 48) + 28))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 48) + 31))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 48) + 34))];
      kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 48) + 37))];
      kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 48) + 40))];
      kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 48) + 43))];
      kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 48) + 46))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(3)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(4)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(5)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(7)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 130))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 153))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 169))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 178))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 185))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 194))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 201))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 226))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 233))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 242))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 249))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 48) + 26))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 48) + 29))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 48) + 32))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 48) + 35))];
      kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 48) + 38))];
      kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 48) + 41))];
      kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 48) + 44))];
      kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 48) + 47))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(3)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(4)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(5)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(7)]));
    }
  }
  compute[((((((int)threadIdx.z) * 196) + (((int)blockIdx.y) * 14)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[(((((((int)threadIdx.z) * 196) + (((int)blockIdx.y) * 14)) + ((int)threadIdx.x)) + 7))] = compute_local[(1)];
}
//grid=(7,2,8),  block=(2,1,4)
int main(int argc, char *argv[]){
    dim3 grid(1,14,1);
    dim3 block(7,1,32);
    int C = atoi(argv[1]);
    int H = atoi(argv[3]);
    int W = atoi(argv[4]);
    int N = atoi(argv[2]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}
