//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[32];
  __shared__ float pad_temp_shared[3072];
  __shared__ float kernel_shared[1536];
  float pad_temp_shared_local[64];
  float kernel_shared_local[2];
  for (int yy_c_init = 0; yy_c_init < 2; ++yy_c_init) {
    for (int xx_c_init = 0; xx_c_init < 8; ++xx_c_init) {
      compute_local[(((yy_c_init * 8) + xx_c_init))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 8) + xx_c_init) + 16))] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    for (int rx_outer = 0; rx_outer < 3; ++rx_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 24; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
        pad_temp_shared[(((((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 48)) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.z) * 3) + ((((((int)threadIdx.y) * 48) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 5)) % 6))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.z) * 3) + ((((((int)threadIdx.y) * 48) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 5)) % 6)) < 225)) && (1 <= (((((int)blockIdx.x) * 32) + rx_outer) + ((((((int)threadIdx.y) * 48) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 31)))) && ((((((int)blockIdx.x) * 32) + rx_outer) + ((((((int)threadIdx.y) * 48) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 31)) < 225)) ? data[(((((((((rc_outer * 802816) + ((((((int)threadIdx.z) * 3) + ((((((int)threadIdx.y) * 48) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 5)) / 6) * 50176)) + (((int)blockIdx.y) * 896)) + ((((((int)threadIdx.z) * 3) + ((((((int)threadIdx.y) * 48) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 5)) % 6) * 224)) + (((int)blockIdx.x) * 32)) + rx_outer) + ((((((int)threadIdx.y) * 48) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 31)) - 225))] : 0.000000e+00f);
      }
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 12; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
        kernel_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((((int)threadIdx.z) * 576) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 * 3)) + rx_outer))];
      }
      __syncthreads();
      for (int rc_inner_outer = 0; rc_inner_outer < 8; ++rc_inner_outer) {
        for (int ry_inner_outer = 0; ry_inner_outer < 3; ++ry_inner_outer) {
          for (int ax1 = 0; ax1 < 2; ++ax1) {
            for (int ax2 = 0; ax2 < 2; ++ax2) {
              for (int ax3 = 0; ax3 < 8; ++ax3) {
                pad_temp_shared_local[((((ax1 * 16) + (ax2 * 8)) + ax3))] = pad_temp_shared[((((((((rc_inner_outer * 384) + (ax1 * 192)) + (((int)threadIdx.y) * 64)) + (ax2 * 32)) + (ry_inner_outer * 32)) + (((int)threadIdx.x) * 8)) + ax3))];
                pad_temp_shared_local[(((((ax1 * 16) + (ax2 * 8)) + ax3) + 32))] = pad_temp_shared[(((((((((rc_inner_outer * 384) + (ax1 * 192)) + (((int)threadIdx.y) * 64)) + (ax2 * 32)) + (ry_inner_outer * 32)) + (((int)threadIdx.x) * 8)) + ax3) + 16))];
              }
            }
          }
          for (int ax11 = 0; ax11 < 2; ++ax11) {
            kernel_shared_local[(ax11)] = kernel_shared[(((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + (ax11 * 3)) + ry_inner_outer))];
          }
          for (int rc_inner_inner = 0; rc_inner_inner < 2; ++rc_inner_inner) {
            for (int yy_c = 0; yy_c < 2; ++yy_c) {
              for (int xx_c = 0; xx_c < 8; ++xx_c) {
                compute_local[(((yy_c * 8) + xx_c))] = (compute_local[(((yy_c * 8) + xx_c))] + (pad_temp_shared_local[((((rc_inner_inner * 16) + (yy_c * 8)) + xx_c))] * kernel_shared_local[(rc_inner_inner)]));
                compute_local[((((yy_c * 8) + xx_c) + 16))] = (compute_local[((((yy_c * 8) + xx_c) + 16))] + (pad_temp_shared_local[(((((rc_inner_inner * 16) + (yy_c * 8)) + xx_c) + 32))] * kernel_shared_local[(rc_inner_inner)]));
              }
            }
          }
        }
      }
    }
  }
  for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 2; ++yy_inner_inner_inner) {
    for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 8; ++xx_inner_inner_inner) {
      compute[((((((((((int)threadIdx.z) * 50176) + (((int)blockIdx.y) * 896)) + (((int)threadIdx.y) * 448)) + (yy_inner_inner_inner * 224)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) * 8)) + xx_inner_inner_inner))] = compute_local[(((yy_inner_inner_inner * 8) + xx_inner_inner_inner))];
      compute[(((((((((((int)threadIdx.z) * 50176) + (((int)blockIdx.y) * 896)) + (((int)threadIdx.y) * 448)) + (yy_inner_inner_inner * 224)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) * 8)) + xx_inner_inner_inner) + 16))] = compute_local[((((yy_inner_inner_inner * 8) + xx_inner_inner_inner) + 16))];
    }
  }
}

int main(int argc, char *argv[]){
    dim3 grid(7,56,1);
    dim3 block(2,2,32);
    int C = atoi(argv[1]);
    int H = atoi(argv[3]);
    int W = atoi(argv[4]);
    int N = atoi(argv[2]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}
