//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;

extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[1792];
  __shared__ float kernel_shared[512];
  float pad_temp_shared_local[64];
  float kernel_shared_local[64];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      for (int rx_outer = 0; rx_outer < 3; ++rx_outer) {
        __syncthreads();
        pad_temp_shared[((((((int)threadIdx.z) * 224) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 8)))] = (((((1 <= (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) % 7) * 8) / 28)) + ry_outer)) && ((((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) % 7) * 8) / 28)) + ry_outer) < 29)) && (1 <= (rx_outer + ((((int)threadIdx.x) * 8) % 28)))) && ((rx_outer + ((((int)threadIdx.x) * 8) % 28)) < 29)) ? data[((((((((((rc_outer * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 1568)) + ((((int)threadIdx.x) / 7) * 784)) + (((int)blockIdx.y) * 56)) + (ry_outer * 28)) + ((((int)threadIdx.x) % 7) * 8)) + rx_outer) - 29))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 224) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 8)) + 1))] = (((((1 <= (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 1) % 56) / 28)) + ry_outer)) && ((((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 1) % 56) / 28)) + ry_outer) < 29)) && (1 <= (rx_outer + (((((int)threadIdx.x) * 8) + 1) % 28)))) && ((rx_outer + (((((int)threadIdx.x) * 8) + 1) % 28)) < 29)) ? data[((((((((((rc_outer * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 8) + 1) / 56) * 784)) + (((int)blockIdx.y) * 56)) + (ry_outer * 28)) + rx_outer) + (((((int)threadIdx.x) * 8) + 1) % 56)) - 29))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 224) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 8)) + 2))] = (((((1 <= (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 2) % 56) / 28)) + ry_outer)) && ((((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 2) % 56) / 28)) + ry_outer) < 29)) && (1 <= (rx_outer + (((((int)threadIdx.x) * 8) + 2) % 28)))) && ((rx_outer + (((((int)threadIdx.x) * 8) + 2) % 28)) < 29)) ? data[((((((((((rc_outer * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 8) + 2) / 56) * 784)) + (((int)blockIdx.y) * 56)) + (ry_outer * 28)) + rx_outer) + (((((int)threadIdx.x) * 8) + 2) % 56)) - 29))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 224) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 8)) + 3))] = (((((1 <= (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 3) % 56) / 28)) + ry_outer)) && ((((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 3) % 56) / 28)) + ry_outer) < 29)) && (1 <= (rx_outer + (((((int)threadIdx.x) * 8) + 3) % 28)))) && ((rx_outer + (((((int)threadIdx.x) * 8) + 3) % 28)) < 29)) ? data[((((((((((rc_outer * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 8) + 3) / 56) * 784)) + (((int)blockIdx.y) * 56)) + (ry_outer * 28)) + rx_outer) + (((((int)threadIdx.x) * 8) + 3) % 56)) - 29))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 224) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 8)) + 4))] = (((((1 <= (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 4) % 56) / 28)) + ry_outer)) && ((((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 4) % 56) / 28)) + ry_outer) < 29)) && (1 <= (rx_outer + (((((int)threadIdx.x) * 8) + 4) % 28)))) && ((rx_outer + (((((int)threadIdx.x) * 8) + 4) % 28)) < 29)) ? data[((((((((((rc_outer * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 8) + 4) / 56) * 784)) + (((int)blockIdx.y) * 56)) + (ry_outer * 28)) + rx_outer) + (((((int)threadIdx.x) * 8) + 4) % 56)) - 29))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 224) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 8)) + 5))] = (((((1 <= (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 5) % 56) / 28)) + ry_outer)) && ((((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 5) % 56) / 28)) + ry_outer) < 29)) && (1 <= (rx_outer + (((((int)threadIdx.x) * 8) + 5) % 28)))) && ((rx_outer + (((((int)threadIdx.x) * 8) + 5) % 28)) < 29)) ? data[((((((((((rc_outer * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 8) + 5) / 56) * 784)) + (((int)blockIdx.y) * 56)) + (ry_outer * 28)) + rx_outer) + (((((int)threadIdx.x) * 8) + 5) % 56)) - 29))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 224) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 8)) + 6))] = (((((1 <= (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 6) % 56) / 28)) + ry_outer)) && ((((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 6) % 56) / 28)) + ry_outer) < 29)) && (1 <= (rx_outer + (((((int)threadIdx.x) * 8) + 6) % 28)))) && ((rx_outer + (((((int)threadIdx.x) * 8) + 6) % 28)) < 29)) ? data[((((((((((rc_outer * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 8) + 6) / 56) * 784)) + (((int)blockIdx.y) * 56)) + (ry_outer * 28)) + rx_outer) + (((((int)threadIdx.x) * 8) + 6) % 56)) - 29))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 224) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 8)) + 7))] = (((((1 <= (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 7) % 56) / 28)) + ry_outer)) && ((((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 8) + 7) % 56) / 28)) + ry_outer) < 29)) && (1 <= (rx_outer + (((((int)threadIdx.x) * 8) + 7) % 28)))) && ((rx_outer + (((((int)threadIdx.x) * 8) + 7) % 28)) < 29)) ? data[((((((((((rc_outer * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 8) + 7) / 56) * 784)) + (((int)blockIdx.y) * 56)) + (ry_outer * 28)) + rx_outer) + (((((int)threadIdx.x) * 8) + 7) % 56)) - 29))] : 0.000000e+00f);
        if ((((((int)threadIdx.z) * 2) + ((((int)threadIdx.x) * 3) >> 5)) + ((int)threadIdx.y)) < 16) {
          if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 3)) < 512) {
            if (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 3)) < 64) {
              if (((int)threadIdx.x) < 11) {
                kernel_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 3)))] = kernel[((((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 1152)) + (((int)threadIdx.y) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 27)) + (ry_outer * 3)) + rx_outer))];
              }
            }
          }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 3) + 1) >> 5)) + ((int)threadIdx.y)) < 16) {
          if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 3)) < 511) {
            if (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 3)) < 63) {
              if (((int)threadIdx.x) < 11) {
                kernel_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 3)) + 1))] = kernel[(((((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 1152)) + (((int)threadIdx.y) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 27)) + (ry_outer * 3)) + rx_outer) + 9))];
              }
            }
          }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 3) + 2) >> 5)) + ((int)threadIdx.y)) < 16) {
          if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 3)) < 510) {
            if (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 3)) < 62) {
              if (((int)threadIdx.x) < 10) {
                kernel_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 3)) + 2))] = kernel[(((((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 1152)) + (((int)threadIdx.y) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 27)) + (ry_outer * 3)) + rx_outer) + 18))];
              }
            }
          }
        }
        __syncthreads();
        pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))];
        pad_temp_shared_local[(32)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 14))];
        pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 56))];
        pad_temp_shared_local[(33)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 70))];
        pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 112))];
        pad_temp_shared_local[(34)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 126))];
        pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 168))];
        pad_temp_shared_local[(35)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 182))];
        pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 224))];
        pad_temp_shared_local[(36)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 238))];
        pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 280))];
        pad_temp_shared_local[(37)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 294))];
        pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 336))];
        pad_temp_shared_local[(38)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 350))];
        pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 392))];
        pad_temp_shared_local[(39)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 406))];
        pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 448))];
        pad_temp_shared_local[(40)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 462))];
        pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 504))];
        pad_temp_shared_local[(41)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 518))];
        pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 560))];
        pad_temp_shared_local[(42)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 574))];
        pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 616))];
        pad_temp_shared_local[(43)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 630))];
        pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 672))];
        pad_temp_shared_local[(44)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 686))];
        pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 728))];
        pad_temp_shared_local[(45)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 742))];
        pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784))];
        pad_temp_shared_local[(46)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 798))];
        pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 840))];
        pad_temp_shared_local[(47)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 854))];
        pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 896))];
        pad_temp_shared_local[(48)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 910))];
        pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 952))];
        pad_temp_shared_local[(49)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 966))];
        pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1008))];
        pad_temp_shared_local[(50)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1022))];
        pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1064))];
        pad_temp_shared_local[(51)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1078))];
        pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1120))];
        pad_temp_shared_local[(52)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1134))];
        pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1176))];
        pad_temp_shared_local[(53)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1190))];
        pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1232))];
        pad_temp_shared_local[(54)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1246))];
        pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1288))];
        pad_temp_shared_local[(55)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1302))];
        pad_temp_shared_local[(24)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1344))];
        pad_temp_shared_local[(56)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1358))];
        pad_temp_shared_local[(25)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1400))];
        pad_temp_shared_local[(57)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1414))];
        pad_temp_shared_local[(26)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1456))];
        pad_temp_shared_local[(58)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1470))];
        pad_temp_shared_local[(27)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1512))];
        pad_temp_shared_local[(59)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1526))];
        pad_temp_shared_local[(28)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568))];
        pad_temp_shared_local[(60)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1582))];
        pad_temp_shared_local[(29)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1624))];
        pad_temp_shared_local[(61)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1638))];
        pad_temp_shared_local[(30)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1680))];
        pad_temp_shared_local[(62)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1694))];
        pad_temp_shared_local[(31)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1736))];
        pad_temp_shared_local[(63)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1750))];
        kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 64))];
        kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 64) + 1))];
        kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 64) + 2))];
        kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 64) + 3))];
        kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 64) + 4))];
        kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 64) + 5))];
        kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 64) + 6))];
        kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 64) + 7))];
        kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 64) + 8))];
        kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 64) + 9))];
        kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 64) + 10))];
        kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 64) + 11))];
        kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 64) + 12))];
        kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 64) + 13))];
        kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 64) + 14))];
        kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 64) + 15))];
        kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 64) + 16))];
        kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 64) + 17))];
        kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 64) + 18))];
        kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 64) + 19))];
        kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 64) + 20))];
        kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 64) + 21))];
        kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 64) + 22))];
        kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 64) + 23))];
        kernel_shared_local[(24)] = kernel_shared[(((((int)threadIdx.z) * 64) + 24))];
        kernel_shared_local[(25)] = kernel_shared[(((((int)threadIdx.z) * 64) + 25))];
        kernel_shared_local[(26)] = kernel_shared[(((((int)threadIdx.z) * 64) + 26))];
        kernel_shared_local[(27)] = kernel_shared[(((((int)threadIdx.z) * 64) + 27))];
        kernel_shared_local[(28)] = kernel_shared[(((((int)threadIdx.z) * 64) + 28))];
        kernel_shared_local[(29)] = kernel_shared[(((((int)threadIdx.z) * 64) + 29))];
        kernel_shared_local[(30)] = kernel_shared[(((((int)threadIdx.z) * 64) + 30))];
        kernel_shared_local[(31)] = kernel_shared[(((((int)threadIdx.z) * 64) + 31))];
        kernel_shared_local[(32)] = kernel_shared[(((((int)threadIdx.z) * 64) + 32))];
        kernel_shared_local[(33)] = kernel_shared[(((((int)threadIdx.z) * 64) + 33))];
        kernel_shared_local[(34)] = kernel_shared[(((((int)threadIdx.z) * 64) + 34))];
        kernel_shared_local[(35)] = kernel_shared[(((((int)threadIdx.z) * 64) + 35))];
        kernel_shared_local[(36)] = kernel_shared[(((((int)threadIdx.z) * 64) + 36))];
        kernel_shared_local[(37)] = kernel_shared[(((((int)threadIdx.z) * 64) + 37))];
        kernel_shared_local[(38)] = kernel_shared[(((((int)threadIdx.z) * 64) + 38))];
        kernel_shared_local[(39)] = kernel_shared[(((((int)threadIdx.z) * 64) + 39))];
        kernel_shared_local[(40)] = kernel_shared[(((((int)threadIdx.z) * 64) + 40))];
        kernel_shared_local[(41)] = kernel_shared[(((((int)threadIdx.z) * 64) + 41))];
        kernel_shared_local[(42)] = kernel_shared[(((((int)threadIdx.z) * 64) + 42))];
        kernel_shared_local[(43)] = kernel_shared[(((((int)threadIdx.z) * 64) + 43))];
        kernel_shared_local[(44)] = kernel_shared[(((((int)threadIdx.z) * 64) + 44))];
        kernel_shared_local[(45)] = kernel_shared[(((((int)threadIdx.z) * 64) + 45))];
        kernel_shared_local[(46)] = kernel_shared[(((((int)threadIdx.z) * 64) + 46))];
        kernel_shared_local[(47)] = kernel_shared[(((((int)threadIdx.z) * 64) + 47))];
        kernel_shared_local[(48)] = kernel_shared[(((((int)threadIdx.z) * 64) + 48))];
        kernel_shared_local[(49)] = kernel_shared[(((((int)threadIdx.z) * 64) + 49))];
        kernel_shared_local[(50)] = kernel_shared[(((((int)threadIdx.z) * 64) + 50))];
        kernel_shared_local[(51)] = kernel_shared[(((((int)threadIdx.z) * 64) + 51))];
        kernel_shared_local[(52)] = kernel_shared[(((((int)threadIdx.z) * 64) + 52))];
        kernel_shared_local[(53)] = kernel_shared[(((((int)threadIdx.z) * 64) + 53))];
        kernel_shared_local[(54)] = kernel_shared[(((((int)threadIdx.z) * 64) + 54))];
        kernel_shared_local[(55)] = kernel_shared[(((((int)threadIdx.z) * 64) + 55))];
        kernel_shared_local[(56)] = kernel_shared[(((((int)threadIdx.z) * 64) + 56))];
        kernel_shared_local[(57)] = kernel_shared[(((((int)threadIdx.z) * 64) + 57))];
        kernel_shared_local[(58)] = kernel_shared[(((((int)threadIdx.z) * 64) + 58))];
        kernel_shared_local[(59)] = kernel_shared[(((((int)threadIdx.z) * 64) + 59))];
        kernel_shared_local[(60)] = kernel_shared[(((((int)threadIdx.z) * 64) + 60))];
        kernel_shared_local[(61)] = kernel_shared[(((((int)threadIdx.z) * 64) + 61))];
        kernel_shared_local[(62)] = kernel_shared[(((((int)threadIdx.z) * 64) + 62))];
        kernel_shared_local[(63)] = kernel_shared[(((((int)threadIdx.z) * 64) + 63))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(0)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(32)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(32)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(1)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(33)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(33)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(2)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(34)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(34)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(3)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(35)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(35)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(4)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(36)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(36)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(5)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(37)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(37)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(6)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(38)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(38)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(7)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(39)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(39)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(8)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(40)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(40)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(9)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(41)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(41)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(10)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(42)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(42)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(11)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(43)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(43)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(12)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(44)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(44)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(13)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(45)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(45)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(14)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(46)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(46)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(15)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(47)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(47)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(16)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(48)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(48)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(17)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(49)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(49)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(18)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(50)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(50)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(51)] * kernel_shared_local[(19)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(51)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(51)] * kernel_shared_local[(51)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(52)] * kernel_shared_local[(20)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(52)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(52)] * kernel_shared_local[(52)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(21)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(53)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(53)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(54)] * kernel_shared_local[(22)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(54)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(54)] * kernel_shared_local[(54)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(55)] * kernel_shared_local[(23)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(55)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(55)] * kernel_shared_local[(55)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(24)] * kernel_shared_local[(24)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(56)] * kernel_shared_local[(24)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(24)] * kernel_shared_local[(56)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(56)] * kernel_shared_local[(56)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(25)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(57)] * kernel_shared_local[(25)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(57)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(57)] * kernel_shared_local[(57)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(26)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(58)] * kernel_shared_local[(26)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(58)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(58)] * kernel_shared_local[(58)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(27)] * kernel_shared_local[(27)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(59)] * kernel_shared_local[(27)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(27)] * kernel_shared_local[(59)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(59)] * kernel_shared_local[(59)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(28)] * kernel_shared_local[(28)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(60)] * kernel_shared_local[(28)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(28)] * kernel_shared_local[(60)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(60)] * kernel_shared_local[(60)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(29)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(61)] * kernel_shared_local[(29)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(61)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(61)] * kernel_shared_local[(61)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(30)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(62)] * kernel_shared_local[(30)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(62)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(62)] * kernel_shared_local[(62)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(31)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(63)] * kernel_shared_local[(31)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(63)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(63)] * kernel_shared_local[(63)]));
      }
    }
  }
  compute[((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 1568)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 1568)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)) + 14))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 1568)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)) + 784))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 1568)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)) + 798))] = compute_local[(3)];
}

//grid=(7,1,16),  block=(2,28,1)
int main(int argc, char *argv[]){
    dim3 grid(1,14,2);
    dim3 block(14,2,8);
    int C = atoi(argv[1]);
    int H = atoi(argv[3]);
    int W = atoi(argv[4]);
    int N = atoi(argv[2]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}
