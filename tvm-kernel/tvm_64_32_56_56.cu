//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[200];
  __shared__ float kernel_shared[288];
  float pad_temp_shared_local[8];
  float kernel_shared_local[12];
  #pragma unroll
  for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
    #pragma unroll
    for (int xx_c_init = 0; xx_c_init < 2; ++xx_c_init) {
      compute_local[(((ff_c_init * 2) + xx_c_init))] = 0.000000e+00f;
      compute_local[((((ff_c_init * 2) + xx_c_init) + 8))] = 0.000000e+00f;
      compute_local[((((ff_c_init * 2) + xx_c_init) + 4))] = 0.000000e+00f;
      compute_local[((((ff_c_init * 2) + xx_c_init) + 12))] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 10)) < 20) {
        if (((((((int)threadIdx.z) * 50) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 200) {
          if ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 50) {
            if (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 7) {
              pad_temp_shared[(((((((int)threadIdx.z) * 50) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 8) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 10)) % 10))) && (((((int)blockIdx.y) * 8) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 10)) % 10)) < 57)) && (1 <= ((((int)blockIdx.x) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 10)))) && (((((int)blockIdx.x) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 10)) < 57)) ? data[((((((((rc_outer * 6272) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 10)) / 10) * 3136)) + (((int)blockIdx.y) * 448)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 10)) % 10) * 56)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 10)) - 57))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 4) + (((((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 9) + ((int)threadIdx.y)) >> 1)) < 16) {
        if ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 9)) + ((int)threadIdx.y)) < 32) {
          if ((((((int)threadIdx.z) * 24) + (((int)threadIdx.y) * 3)) + (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 3)) < 96) {
            if (((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 9)) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 288) {
              if ((((((int)threadIdx.y) * 9) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 72) {
                if (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 9) {
                  kernel_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 9)) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[((((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 2304)) + ((((int)threadIdx.y) >> 1) * 576)) + (rc_outer * 18)) + ((((int)threadIdx.y) & 1) * 9)) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))];
                }
              }
            }
          }
        }
      }
    }
    __syncthreads();
    #pragma unroll
    for (int rc_inner_outer = 0; rc_inner_outer < 2; ++rc_inner_outer) {
      #pragma unroll
      for (int ry_inner_outer = 0; ry_inner_outer < 3; ++ry_inner_outer) {
        #pragma unroll
        for (int ax3 = 0; ax3 < 4; ++ax3) {
          pad_temp_shared_local[(ax3)] = pad_temp_shared[((((((rc_inner_outer * 100) + (((int)threadIdx.y) * 10)) + (ry_inner_outer * 10)) + (((int)threadIdx.x) * 2)) + ax3))];
          pad_temp_shared_local[((ax3 + 4))] = pad_temp_shared[(((((((rc_inner_outer * 100) + (((int)threadIdx.y) * 10)) + (ry_inner_outer * 10)) + (((int)threadIdx.x) * 2)) + ax3) + 4))];
        }
        #pragma unroll
        for (int ax0 = 0; ax0 < 2; ++ax0) {
          #pragma unroll
          for (int ax31 = 0; ax31 < 3; ++ax31) {
            kernel_shared_local[(((ax0 * 3) + ax31))] = kernel_shared[((((((((int)threadIdx.z) * 36) + (ax0 * 18)) + (rc_inner_outer * 9)) + (ry_inner_outer * 3)) + ax31))];
            kernel_shared_local[((((ax0 * 3) + ax31) + 6))] = kernel_shared[(((((((((int)threadIdx.z) * 36) + (ax0 * 18)) + (rc_inner_outer * 9)) + (ry_inner_outer * 3)) + ax31) + 144))];
          }
        }
        #pragma unroll
        for (int rx_inner_inner = 0; rx_inner_inner < 3; ++rx_inner_inner) {
          #pragma unroll
          for (int ff_c = 0; ff_c < 2; ++ff_c) {
            #pragma unroll
            for (int xx_c = 0; xx_c < 2; ++xx_c) {
              compute_local[(((ff_c * 2) + xx_c))] = (compute_local[(((ff_c * 2) + xx_c))] + (pad_temp_shared_local[((xx_c + rx_inner_inner))] * kernel_shared_local[(((ff_c * 3) + rx_inner_inner))]));
              compute_local[((((ff_c * 2) + xx_c) + 8))] = (compute_local[((((ff_c * 2) + xx_c) + 8))] + (pad_temp_shared_local[((xx_c + rx_inner_inner))] * kernel_shared_local[((((ff_c * 3) + rx_inner_inner) + 6))]));
              compute_local[((((ff_c * 2) + xx_c) + 4))] = (compute_local[((((ff_c * 2) + xx_c) + 4))] + (pad_temp_shared_local[(((xx_c + rx_inner_inner) + 4))] * kernel_shared_local[(((ff_c * 3) + rx_inner_inner))]));
              compute_local[((((ff_c * 2) + xx_c) + 12))] = (compute_local[((((ff_c * 2) + xx_c) + 12))] + (pad_temp_shared_local[(((xx_c + rx_inner_inner) + 4))] * kernel_shared_local[((((ff_c * 3) + rx_inner_inner) + 6))]));
            }
          }
        }
      }
    }
  }
  #pragma unroll
  for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2; ++ff_inner_inner_inner) {
    #pragma unroll
    for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 2; ++xx_inner_inner_inner) {
      compute[(((((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner))] = compute_local[(((ff_inner_inner_inner * 2) + xx_inner_inner_inner))];
      compute[((((((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 25088))] = compute_local[((((ff_inner_inner_inner * 2) + xx_inner_inner_inner) + 8))];
      compute[((((((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 4))] = compute_local[((((ff_inner_inner_inner * 2) + xx_inner_inner_inner) + 4))];
      compute[((((((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 25092))] = compute_local[((((ff_inner_inner_inner * 2) + xx_inner_inner_inner) + 12))];
    }
  }
}

//grid=(56,56,32),  block=(1,1,1)
int main(int argc, char *argv[]){
    dim3 grid(7,7,2);
    dim3 block(2,8,4);
    int C = atoi(argv[1]);
    int H = atoi(argv[3]);
    int W = atoi(argv[4]);
    int N = atoi(argv[2]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}
