//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[1];
  __shared__ float kernel_shared[1];
  float pad_temp_shared_local[1];
  float kernel_shared_local[1];
  compute_local[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 64; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      for (int rx_outer = 0; rx_outer < 3; ++rx_outer) {
        pad_temp_shared[(0)] = (((((1 <= (((int)blockIdx.y) + ry_outer)) && ((((int)blockIdx.y) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 49) + (((int)blockIdx.y) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
        kernel_shared[(0)] = kernel[(((((((int)blockIdx.z) * 576) + (rc_outer * 9)) + (ry_outer * 3)) + rx_outer))];
        pad_temp_shared_local[(0)] = pad_temp_shared[(0)];
        kernel_shared_local[(0)] = kernel_shared[(0)];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      }
    }
  }
  compute[((((((int)blockIdx.z) * 49) + (((int)blockIdx.y) * 7)) + ((int)blockIdx.x)))] = compute_local[(0)];
}

//grid=(1,1,2),  block=(7,7,4)
int main(int argc, char *argv[]){
    dim3 grid(7,7,32);
    dim3 block(1,1,1);
    int C = atoi(argv[1]);
    int H = atoi(argv[3]);
    int W = atoi(argv[4]);
    int N = atoi(argv[2]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}
