//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
//grid=(1,7,16),  block=(7,1,2)
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[2592];
  __shared__ float kernel_shared[4608];
  float pad_temp_shared_local[48];
  float kernel_shared_local[192];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 3; ++rc_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 14; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 93) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 81)) < 32) {
        if (((((int)threadIdx.z) * 72) + ((((((int)threadIdx.y) * 93) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 9)) < 288) {
          if (((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 93)) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 2592) {
            if ((((((int)threadIdx.y) * 93) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 648) {
              if (((((int)threadIdx.x) * 14) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 93) {
                pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 93)) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((9 <= ((((((int)threadIdx.y) * 93) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 81)) && (((((((int)threadIdx.y) * 93) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 93) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 9))) && (((((((int)threadIdx.y) * 93) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 9) < 8)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + (((((((int)threadIdx.y) * 93) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 81) * 49)) + ((((((((int)threadIdx.y) * 93) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 93) + (((int)threadIdx.x) * 14)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 9)) - 8))] : 0.000000e+00f);
              }
            }
          }
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 24; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 8)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 96)) < 16) {
        if (((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 8)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 3)) < 512) {
          if (((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 55)) + (((int)threadIdx.x) * 8)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) < 1536) {
            if (((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 4608) {
              if ((((((int)threadIdx.y) * 165) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 1152) {
                if (((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 165) {
                  if ((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 4)) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 8)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 96)) < 64) {
                    kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 3456)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 8)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 96) * 864)) + (rc_outer * 288)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 8)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) % 96) * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 % 3)))];
                  }
                }
              }
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 2; ++rc_inner_outer) {
      for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
        #pragma unroll
        for (int ax1 = 0; ax1 < 16; ++ax1) {
          #pragma unroll
          for (int ax2 = 0; ax2 < 3; ++ax2) {
            pad_temp_shared_local[(((ax1 * 3) + ax2))] = pad_temp_shared[(((((((rc_inner_outer * 1296) + (ax1 * 81)) + (ax2 * 9)) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + rx_inner_outer))];
          }
        }
        #pragma unroll
        for (int ax11 = 0; ax11 < 16; ++ax11) {
          #pragma unroll
          for (int ax21 = 0; ax21 < 3; ++ax21) {
            kernel_shared_local[(((ax11 * 3) + ax21))] = kernel_shared[((((((((int)threadIdx.z) * 288) + (rc_inner_outer * 144)) + (ax11 * 9)) + (ax21 * 3)) + rx_inner_outer))];
            kernel_shared_local[((((ax11 * 3) + ax21) + 48))] = kernel_shared[(((((((((int)threadIdx.z) * 288) + (rc_inner_outer * 144)) + (ax11 * 9)) + (ax21 * 3)) + rx_inner_outer) + 1152))];
            kernel_shared_local[((((ax11 * 3) + ax21) + 96))] = kernel_shared[(((((((((int)threadIdx.z) * 288) + (rc_inner_outer * 144)) + (ax11 * 9)) + (ax21 * 3)) + rx_inner_outer) + 2304))];
            kernel_shared_local[((((ax11 * 3) + ax21) + 144))] = kernel_shared[(((((((((int)threadIdx.z) * 288) + (rc_inner_outer * 144)) + (ax11 * 9)) + (ax21 * 3)) + rx_inner_outer) + 3456))];
          }
        }
        #pragma unroll
        for (int rc_inner_inner = 0; rc_inner_inner < 16; ++rc_inner_inner) {
          #pragma unroll
          for (int ry_inner_inner = 0; ry_inner_inner < 3; ++ry_inner_inner) {
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))] * kernel_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))] * kernel_shared_local[((((rc_inner_inner * 3) + ry_inner_inner) + 48))]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))] * kernel_shared_local[((((rc_inner_inner * 3) + ry_inner_inner) + 96))]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))] * kernel_shared_local[((((rc_inner_inner * 3) + ry_inner_inner) + 144))]));
          }
        }
      }
    }
  }
  compute[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196))] = compute_local[(1)];
  compute[((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392))] = compute_local[(2)];
  compute[((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 588))] = compute_local[(3)];
}

int main(int argc, char *argv[]){
    dim3 grid(1,1,4);
    dim3 block(7,7,4);
    int C = atoi(argv[1]);
    int H = atoi(argv[3]);
    int W = atoi(argv[4]);
    int N = atoi(argv[2]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}
